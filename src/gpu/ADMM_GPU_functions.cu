/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 3
 * of the programming guide with some additions like error checking.
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_fp16.h>

////////////////////////////////////////////////////////////////////////////////////////////////////////

#define SWAP_des(x,y) sort2_swap_des(&d##x, &d##y, &p##x, &p##y)
__device__ void sort2_swap_des(float* dx, float* dy, int* px, int* py)
{
	const auto Dx = *dx, Dy = (*dy);
	const auto Px = *px, Py = (*py);
	const bool test = (Dx > Dy);
	(*dx) = fmaxf(Dx,Dy);
	(*dy) = fminf(Dx,Dy);
	(*px) = test ? Px : Py;
	(*py) = test ? Py : Px;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////

#define SWAP_asc(x,y) sort2_swap_asc(&d##x, &d##y, &p##x, &p##y)
__device__ void sort2_swap_asc(float* dx, float* dy, int* px, int* py)
{
	const auto Dx = *dx, Dy = (*dy);
	const auto Px = *px, Py = (*py);
	const bool test = (Dx < Dy);
	(*dx) = fminf(Dx,Dy);
	(*dy) = fmaxf(Dx,Dy);
	(*px) = test ? Px : Py;
	(*py) = test ? Py : Px;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ void sort6_swap(float d[6], int p[6])
{
	#define SWAP SWAP_des
    auto p0 = 0;    auto p1 = 1;    auto p2 = 2;
    auto p3 = 3;    auto p4 = 4;    auto p5 = 5;
    float d0 = d[0]; float d1 = d[1]; float d2 = d[2];
    float d3 = d[3]; float d4 = d[4]; float d5 = d[5];
    SWAP(1, 2); SWAP(0, 2); SWAP(0, 1); SWAP(4, 5);
    SWAP(3, 5); SWAP(3, 4); SWAP(0, 3); SWAP(1, 4);
    SWAP(2, 5); SWAP(2, 4); SWAP(1, 3); SWAP(2, 3);
    d[0] = d0; d[1] = d1; d[2] = d2;
    d[3] = d3; d[4] = d4; d[5] = d5;
    p[0] = p0; p[1] = p1; p[2] = p2;
    p[3] = p3; p[4] = p4; p[5] = p5;
	#undef SWAP
}

////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ void sort6_swap(float illr[6], float rllr[6], int ipos[6], int rpos[6])
{
	#define SWAP SWAP_asc
    auto  p0 = ipos[0]; auto  p1 = ipos[1]; auto  p2 = ipos[2];
    auto  p3 = ipos[3]; auto  p4 = ipos[4]; auto  p5 = ipos[5];
    float d0 = illr[0]; float d1 = illr[1]; float d2 = illr[2];
    float d3 = illr[3]; float d4 = illr[4]; float d5 = illr[5];
    SWAP(1, 2); SWAP(0, 2); SWAP(0, 1); SWAP(4, 5);
    SWAP(3, 5); SWAP(3, 4); SWAP(0, 3); SWAP(1, 4);
    SWAP(2, 5); SWAP(2, 4); SWAP(1, 3); SWAP(2, 3);
    rllr[0] = d0; rllr[1] = d1; rllr[2] = d2;
    rllr[3] = d3; rllr[4] = d4; rllr[5] = d5;
    rpos[0] = p0; rpos[1] = p1; rpos[2] = p2;
    rpos[3] = p3; rpos[4] = p4; rpos[5] = p5;
	#undef SWAP
}

////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ void sort6_rank_order_reg(float llr[ ], int pos[ ])
{
	const float x0 = llr[0]; const float x1 = llr[1]; const float x2 = llr[2];
    const float x3 = llr[3]; const float x4 = llr[4]; const float x5 = llr[5];
    const int   o0 = (x0< x1) + (x0< x2) + (x0< x3) + (x0< x4) + (x0<x5);
    const int   o1 = (x1<=x0) + (x1< x2) + (x1< x3) + (x1< x4) + (x1<x5);
    const int   o2 = (x2<=x0) + (x2<=x1) + (x2< x3) + (x2< x4) + (x2<x5);
    const int   o3 = (x3<=x0) + (x3<=x1) + (x3<=x2) + (x3< x4) + (x3<x5);
    const int   o4 = (x4<=x0) + (x4<=x1) + (x4<=x2) + (x4<=x3) + (x4<x5);
    const int   o5 = 15 - (o0 + o1 + o2 + o3 + o4);
    llr[o0]=x0; llr[o1]=x1; llr[o2]=x2; llr[o3]=x3; llr[o4]=x4; llr[o5]=x5;
    pos[o0]= 0; pos[o1]= 1; pos[o2]= 2; pos[o3]= 3; pos[o4]= 4; pos[o5]= 5;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ void sort6_rank_order_reg_modif(float illr[ ], float rllr[ ], int ipos[ ], int rpos[ ])
{
	const float x0 = illr[0], x1 = illr[1], x2 = illr[2];
	const float x3 = illr[3], x4 = illr[4], x5 = illr[5];
	const int   o0 = (x0> x1) + (x0> x2) + (x0> x3) + (x0> x4) + (x0>x5);
	const int   o1 = (x1>=x0) + (x1> x2) + (x1> x3) + (x1> x4) + (x1>x5);
	const int   o2 = (x2>=x0) + (x2>=x1) + (x2> x3) + (x2> x4) + (x2>x5);
	const int   o3 = (x3>=x0) + (x3>=x1) + (x3>=x2) + (x3> x4) + (x3>x5);
	const int   o4 = (x4>=x0) + (x4>=x1) + (x4>=x2) + (x4>=x3) + (x4>x5);
	const int   o5 = 15 - (o0 + o1 + o2 + o3 + o4);
	rllr[o0]=x0;      rllr[o1]=x1;      rllr[o2]=x2;      rllr[o3]=x3;      rllr[o4]=x4;      rllr[o5]=x5;
	rpos[o0]=ipos[0]; rpos[o1]=ipos[1]; rpos[o2]=ipos[2]; rpos[o3]=ipos[3]; rpos[o4]=ipos[4]; rpos[o5]=ipos[5];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////

__shared__ int sdata[128*6]; // > 512

////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ void projection_deg6(float llr[], float results[])
{
	const int length = 6;
	bool finished    = false;


	/////////////////////////////////////////////////////////////////////////////////////////////////////////
	int AllZero = (llr[0] <= 0);
	int AllOne  = (llr[0] <= 0);

	#pragma unroll
	for(int i = 1; i < length; i++)
	{
		AllZero = AllZero + (llr[i] <= 0);
		AllOne  = AllOne  + (llr[i] <= 0);
	}

	/////////////////////////////////////////////////////////////////////////////////////////////////////////

	finished = (AllZero == length);

    __syncthreads( );

    /////////////////////////////////////////////////////////////////////////////////////////////////////////

    bool test = (finished == false) && (AllOne == length) && ((length&0x01) == 0);
	#pragma unroll
	for(int i = 0; i < length; i++)
		results[i] = (test == true) ? 1.0f : 0.0f;
	finished = finished | test;

    __syncthreads( );

	/////////////////////////////////////////////////////////////////////////////////////////////////////////
	float constituent = 0;
	float llrClip[6];
	int   zSorti[6] = {0, 1, 2, 3, 4, 5};

	sort6_swap(llr, zSorti);

	#pragma unroll
	for(int i = 0; i < length; i++)// project on the [0,1]^d cube
	{
		const float vMax = fminf(fmaxf(llr[i], 0.0f), 1.0f);
		llrClip[i]       = vMax;
		constituent     += vMax;
	}

	int r = (int)constituent;
    r     = r - (r & 0x01);

	float sum_Clip = llrClip[0];
	for(int i = 1; i < length; i++)
	{
		sum_Clip += (i <  r+1) ? llrClip[i] : -llrClip[i];
	}

	// affectation conditionnelle des resultats
	bool valid = ( finished == false ) && (sum_Clip <= r);
	#pragma unroll
	for(int i = 0; i < length; i++)
		results[zSorti[i]] = (valid == true) ? llrClip[i] : results[zSorti[i]];
	finished = finished || valid;

	/////////////////////////////////////////////////////////////////////////////////////////////////////////

    __syncthreads();

	/////////////////////////////////////////////////////////////////////////////////////////////////////////

    if( finished == false )
	{
		float beta     = 0;
		float beta_max = (r + 2 <= length) ? (llr[r] - llr[r+1])/2 : llr[r]; // assign beta_max

	    // sorting zBetaRep
		int   zSorti_m[6] = {0, 1, 2, 3, 4, 5};
		float T_in[6];
	    float T_out[6];
	    int   order_out[6];

		#pragma unroll
		for(int i = 0; i < length; i++)
	        T_in[i] = (i < r+1) ? llr[i] - 1.0f : -llr[i];

		sort6_rank_order_reg_modif (T_in, T_out, zSorti_m, order_out);

		int clip_idx  = -1;
		int zero_idx  =  0;
		int idx_start =  0;
		int idx_end   = -1;

		#pragma unroll 6
		for(int i = 0; i < length; i++)
		{
			clip_idx  += (llr[i]   >     1.0f);
			zero_idx  += (llr[i]   >= -1e-10f);
			idx_start += (T_out[i] <   1e-10f);
			idx_end   += (T_out[i] < beta_max);
		}

		float active_sum = 0;

		#pragma unroll 6
		for(int i = 0;i < length; i++)
		{
			active_sum += (i > clip_idx && i <= r      ) ? llr[i] : 0.0f;
			active_sum -= (i > r        && i < zero_idx) ? llr[i] : 0.0f;
		}

		float total_sum           = active_sum + clip_idx + 1;
		int previous_clip_idx     = clip_idx;
		int previous_zero_idx     = zero_idx;
		float previous_active_sum = active_sum;
		bool change_pre           = false;

		for(int i = idx_start; i <= idx_end; i++)// pour tous les beta entre 0 et beta_max
		{
			if(change_pre)
			{
				// save previous things
				previous_clip_idx   = clip_idx;
				previous_zero_idx   = zero_idx;
				previous_active_sum = active_sum;
			}
			change_pre = false;

			beta = T_out[i];
			clip_idx   -= (order_out[i] <= r);
			zero_idx   += (order_out[i] >  r);
			active_sum += (order_out[i] <= r) ? llr[order_out[i]] : -llr[order_out[i]];

			if (i < length - 1)
			{
				if (beta != T_out[i+1])
				{
					total_sum  = (clip_idx + 1) + active_sum - beta * (zero_idx - clip_idx - 1);
					change_pre = true;
					if(total_sum < r)
						break;
				}

			}
			else if (i == length - 1)
			{
				total_sum  = (clip_idx + 1)  + active_sum - beta * (zero_idx - clip_idx - 1);
				change_pre = true;
			}
		}

		clip_idx   = (total_sum > r) ? clip_idx   : previous_clip_idx;
		active_sum = (total_sum > r) ? active_sum : previous_active_sum;
		zero_idx   = (total_sum > r) ? zero_idx   : previous_zero_idx;
		beta       = -(r - clip_idx - 1 - active_sum)/(zero_idx - clip_idx - 1);

		#pragma unroll 6
		for(int i = 0; i < length; i++)
		{
			const float vA = llr[i];
			const float vD = (i <= r) ? vA - beta : vA + beta;
			results[zSorti[i]] = fminf(fmaxf(vD, 0.0f), 1.0f);
		}
	}

	/////////////////////////////////////////////////////////////////////////////////////////////////////////

    __syncthreads();

	/////////////////////////////////////////////////////////////////////////////////////////////////////////
}



/////////////////////////////////////////////////////////////////////////////////////////////////////////



#define FLOAT2

__global__ void ADMM_InitArrays(float* LZr, int N)
{
//    t.x = 0.00f;
//    t.y = 0.50f;
//	__half  t1 = __float2half (  );
//	__half  t2 = __float2half ( 0.00f );
//  __half2 t3 =__halves2half2 ( t1, t2 );
//  float2 	__half22float2 ( const __half2 a )
    // __high2float
    // __low2float
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
#ifdef FLOAT2
    	float2* ptr = reinterpret_cast<float2*>(LZr);
    	ptr[i]      = make_float2(0.00f, 0.50f);
#else
        Lambda  [i] = 0.00f;
        zReplica[i] = 0.50f;
#endif
    }
}


/////////////////////////////////////////////////////////////////////////////////////////////////////////



__global__ void ADMM_ScaleLLRs(float* LLRs, int N)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
    	const float mu = 3.0f;
    	LLRs[i] = LLRs[i] / mu;
    }
}


/////////////////////////////////////////////////////////////////////////////////////////////////////////


__global__ void ADMM_VN_kernel_deg3(
	const float* _LogLikelihoodRatio, float* OutputFromDecoder, float* LZr, const unsigned int *t_row, int N)
{
    const int i             = blockDim.x * blockIdx.x + threadIdx.x;
	constexpr float mu      = 3.0f;
	constexpr float  alpha  = 0.8;
	constexpr float _amu_   = alpha / mu;
	constexpr float _2_amu_ = _amu_+ _amu_;
    constexpr float factor  = 1.0f / (3.0f - _2_amu_);
    const int   degVn       = 3;

    if (i < N){
        float temp                  = -_LogLikelihoodRatio[i]; // <= OK
        const int frame_offset      = (i%2640);
        const int num_trame         = (i/2640);
        const ushort4  off          = reinterpret_cast<ushort4*>((unsigned int *)t_row)[ frame_offset ];
        const unsigned short tab[4] = {off.x, off.y, off.z, off.w};

        #pragma unroll 3
        for(int k = 0; k < degVn; k++)
        {
//        	const int pos = 3 * i + k;
        	const int off = tab[k];//t_row[ pos ];
#ifdef FLOAT2
        	const float2* ptr = reinterpret_cast<float2*>(LZr);
        	const float2 data = ptr[ (7920 * num_trame) + off ];
            temp       += (data.y + data.x);
#else
            temp       += ( zReplica[ off ] + Lambda[ off ] );
#endif
        }
        const float xx       = (temp  -  _amu_) * factor;
        OutputFromDecoder[i] = fmaxf(fminf(xx, 1.0f), 0.0f);
    }
}



/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////



__global__ void ADMM_CN_kernel_deg6(
	const float *OutputFromDecoder, float *LZr, const unsigned int *t_col1, int *cn_synrome, int N)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x; // NUMERO DU CHECK NODE A CALCULER
	const float rho      = 1.9f;
	const float un_m_rho = 1.0f - rho;
	const int   degCn    = 6;
    float v_proj[6], ztemp [6];

    if (i < N){
        const int frame_offset = i%1320;
        const int trame_start  = 2640 * (i/1320);

    	int syndrom = 0;
        #pragma unroll
        for(int k = 0; k < degCn; k++)
        {
            const int ind      = degCn * i + k;
            const int offset   = t_col1[ degCn * frame_offset + k ];
            const float xpred  = OutputFromDecoder[ trame_start + offset ];
            syndrom           += (xpred > 0.5);
            reinterpret_cast<float*>(sdata)[threadIdx.x + 128 * k] = xpred;
#ifdef FLOAT2
        	const float2* ptr = reinterpret_cast<float2*>(LZr);
        	const float2 data = ptr[ ind ];
            v_proj[k]         = (rho * xpred) + (un_m_rho * data.y) - data.x;
#else
            v_proj[k]         = (rho * xpred) + (un_m_rho * zReplica[ind]) - Lambda[ind];
#endif
        }
        cn_synrome[i] = syndrom & 0x01;

        projection_deg6(v_proj, ztemp);

        #pragma unroll
        for(int k = 0; k < degCn; k++)
        {
            const int ind     = degCn * i + k;
            const float xpred = reinterpret_cast<float*>(sdata)[threadIdx.x + 128 * k];
#ifdef FLOAT2
        	float2* ptr = reinterpret_cast<float2*>(LZr);
            float2 data = ptr[ ind ];
            float x     = data.x + (rho * (ztemp[k] - xpred) + un_m_rho * (ztemp[k] - data.y));
            ptr[ ind ]  = make_float2(x, ztemp[k]);
#else
            Lambda[ind]    = Lambda[ind] + (rho * (ztemp[k] - xpred) + un_m_rho * (ztemp[k] - zReplica[ind]));
            zReplica[ind]  = ztemp[k];
#endif
        }
    }
}


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


__global__ void ADMM_HardDecision(
		float* OutputFromDecoder, int* HardDecision, int N
		)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
        HardDecision[i] = floorf(OutputFromDecoder[i] + 0.50f);
    }
}


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


__global__ void reduce(int *g_idata, unsigned int n)
{
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid      =                               threadIdx.x;
    unsigned int i        = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    unsigned int gridSize = blockDim.x * 2 * gridDim.x;

    int mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        mySum += g_idata[i];
        // ensure we don't read out of bounds
        if (i + blockDim.x < n)
            mySum += g_idata[i+blockDim.x];
        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    if (blockDim.x >= 1024) { if (tid < 512) { sdata[tid] = mySum = mySum + sdata[tid + 512]; } __syncthreads(); }
    if (blockDim.x >=  512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
    if (blockDim.x >=  256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
    if (blockDim.x >=  128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }

    // avoid bank conflict
    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile int* smem = sdata;
        if (blockDim.x >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; }
        if (blockDim.x >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; }
        if (blockDim.x >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; }
        if (blockDim.x >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; }
        if (blockDim.x >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; }
        if (blockDim.x >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; }
    }

    // write result for this block to global mem
    if (tid == 0)
    	g_idata[blockIdx.x] = sdata[0];
}



////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void ADMM_InitArrays_16b(float* LZr, int N)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
    	__half   t1  = __float2half  ( 0.00f  ); // Lambda
    	__half   t2  = __float2half  ( 0.50f  ); // zReplica
    	__half2* ptr = reinterpret_cast<__half2*>(LZr);
    	ptr[i]       = __halves2half2( t1, t2 );
    }
}

__global__ void ADMM_VN_kernel_deg3_16b(
	const float* _LogLikelihoodRatio, float* OutputFromDecoder, float* LZr, const unsigned int *t_row, int N)
{
    const int i             = blockDim.x * blockIdx.x + threadIdx.x;
	constexpr float mu      = 3.0f;
	constexpr float  alpha  = 0.8;
	constexpr float _amu_   = alpha / mu;
	constexpr float _2_amu_ = _amu_+ _amu_;
    constexpr float factor  = 1.0f / (3.0f - _2_amu_);
    const int   degVn       = 3;
	const __half2* ptr      = reinterpret_cast<__half2*>(LZr);

    if (i < N){
        float temp                  = -_LogLikelihoodRatio[i];
        const int frame_offset      = (i%2640);
        const int num_trame         = (i/2640);
        const ushort4  off          = reinterpret_cast<ushort4*>((unsigned int *)t_row)[ frame_offset ];
        const unsigned short tab[4] = {off.x, off.y, off.z, off.w};

        #pragma unroll 3
        for(int k = 0; k < degVn; k++)
        {
        	const int off = tab[k];
        	const __half2 data = ptr[ (7920 * num_trame) + off ];
            temp              += ( __high2float(data) + __low2float(data) );
        }
        const float xx       = (temp  -  _amu_) * factor;
        OutputFromDecoder[i] = fmaxf(fminf(xx, 1.0f), 0.0f);
    }
}

__global__ void ADMM_CN_kernel_deg6_16b(
	const float *OutputFromDecoder, float *Lzr, const unsigned int *t_col1, int *cn_synrome, int N)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x; // NUMERO DU CHECK NODE A CALCULER
    constexpr float rho      = 1.9f;
    constexpr float un_m_rho = 1.0f - rho;
    constexpr int   degCn    = 6;
    float v_proj[6];
    float ztemp [6];
    __half2* ptr = reinterpret_cast<__half2*>(Lzr);
    float*   PTR = reinterpret_cast<float*>(sdata);

    if (i < N){
        const int frame_offset = i%1320;
        const int trame_start  = 2640 * (i/1320);

    	int syndrom = 0;

    	#pragma unroll 6
        for(int k = 0; k < degCn; k++)
        {
            const int ind      = degCn * i + k;
            const int offset   = t_col1[ degCn * frame_offset + k ];
            const float xpred  = OutputFromDecoder[ trame_start + offset ];
            syndrom           += (xpred > 0.5);

            PTR[threadIdx.x + 128 * k] = xpred;
        	const __half2 data         = ptr[ ind ];
        	v_proj[k]                  = (rho * xpred) + (un_m_rho * __high2float(data)) - __low2float(data);
        }
        cn_synrome[i] = syndrom & 0x01;

        projection_deg6(v_proj, ztemp);

        #pragma unroll 6
        for(int k = 0; k < degCn; k++)
        {
            const int ind      = degCn * i + k;
            const float  xpred = PTR[threadIdx.x + 128 * k];
            const __half2 data = ptr[ ind ];
            float x            = __low2float(data) + (rho * (ztemp[k] - xpred) + un_m_rho * (ztemp[k] - __high2float(data)));
            ptr[ ind ]         = __halves2half2( __float2half(x), __float2half(ztemp[k]) );
        }
    }
}
